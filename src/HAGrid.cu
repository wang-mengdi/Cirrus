#include "HAGrid.h"


void CheckCudaError(const std::string& message) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << fmt::format("CUDA error at {}: {}", message, hipGetErrorString(err)) << std::endl;
    }
}